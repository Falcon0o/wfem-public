#include "base/assert.h"
#include "geom/linear_bvh.cuh"

#include <hip/hip_cooperative_groups.h>

#include <bit>
#include "parallel/cuda/algorithm/radix_sort.cuh"
// #include <hip/hip_runtime_api.h>

#include "parallel/cuda/runtime.hpp"

// #include "parallel/cuda/kernel_function.hpp"
// #include "parallel/cuda/device_properties.hpp"

#include "parallel/cuda/deduce_block_size.hpp"
